#include "hip/hip_runtime.h"
#include "utils/cuda.hpp"
#include "sph_scene.hpp"

//#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>

#include <thrust/sort.h>

using namespace px;

struct CudaSPHBuffer
{
    unsigned int capacity = 0;

    // because of the limit of max threads on a GPU
    // multiple streams cannot bring observable improvement
    static constexpr int n_streams = 0;
    hipStream_t *streams = nullptr;

    float *mass = nullptr;
    float *rest_density = nullptr;
    float *coef_viscosity = nullptr;
    float3 *velocity = nullptr;         // velocity
    float3 *velocity_h = nullptr;       // half velocity
    float3 *acceleration = nullptr;     // acceleration
    float *density = nullptr;           // density
    float *pressure = nullptr;          // pressure
    int *grid_index = nullptr;          // index of the grid where the particle is
    int3 *cell_index = nullptr;         // x, y, z component of the grid index
    int *bucket = nullptr;         //
    int *bucket_start = nullptr;   // first index of the particles in a grid, ordered in the sorted index array
    int *bucket_map = nullptr;          // sorted index -> original index in the given velocity and position array
    int *grid_counter = nullptr;        // grid counter

    explicit CudaSPHBuffer() {}
} cuda_buffer;

__global__
void cudaSPHPosition2Grid(const float3 *__restrict__ pos,
                          int *__restrict__ grid_index, int3 *__restrict__ cell_index,
                          const float3 lower_boundary, const float smooth_len, const int3 resolution,
                          const unsigned int n, const int offset)
{
    PX_CUDA_LOOP(id, n)
    {
        auto i = id + offset;
        auto p = pos[i];
        auto x = static_cast<int>((p.x - lower_boundary.x) / smooth_len);
        if (x < 0) x = 0;
        else if (x > resolution.x-1) x = resolution.x-1;
        auto y = static_cast<int>((p.y - lower_boundary.y) / smooth_len);
        if (y < 0) y = 0;
        else if (y > resolution.y-1) y = resolution.y-1;
        auto z = static_cast<int>((p.z - lower_boundary.z) / smooth_len);
        if (z < 0) z = 0;
        else if (z > resolution.z-1) z = resolution.z-1;

        cell_index[i].x = x;
        cell_index[i].y = y;
        cell_index[i].z = z;
        grid_index[i] = (x*resolution.y + y)*resolution.z + z;
    }
}
__global__
void cudaSPHClearCounter(int *__restrict__ grid_counter, const unsigned int n_grids)
{
    PX_CUDA_LOOP(i, n_grids)
    {
        grid_counter[i] = 0;
    }
}
__global__
void cudaSPHGridCount(const int *__restrict__ grid_index, int *__restrict__ grid_counter, const unsigned int n)
{
    PX_CUDA_LOOP(i, n)
    {
        atomicAdd(&grid_counter[grid_index[i]], 1);
    }
}
__global__
void cudaSPHSort(int *__restrict__ pref_sum, const int *__restrict__ grid_index,
                 int *__restrict__ bucket, int *__restrict__ bucket_map,
                 const unsigned int n)
{
    PX_CUDA_LOOP(i, n)
    {
        auto index = grid_index[i];
        auto new_index = atomicAdd(&pref_sum[index], 1);
        bucket[new_index] = index;
        bucket_map[new_index] = i;
    }
}
__global__
void cudaSPHMakeBins(const int *__restrict__ bucket,
                     int *__restrict__ bucket_start,
                     const unsigned int n_grids,
                     const unsigned int n)
{
    PX_CUDA_LOOP(new_i, n)
    {
        auto cur_bucket = bucket[new_i];
        auto prev_bucket = new_i > 0 ? bucket[new_i-1] : 0;
        if (prev_bucket != cur_bucket)
        {
            for (int j = prev_bucket; j < cur_bucket; j++)
                bucket_start[j] = new_i;
        }
        if (new_i == n - 1)
        {
            for (int j = cur_bucket; j < n_grids; j++)
                bucket_start[j] = n;
        }
    }
}
__global__
void cudaSPHDensityAndPressure(const float3 *__restrict__ pos, const float *mass, const float *rest_density,
                               const int* __restrict__ bucket_map, const int *__restrict__ bucket_start,
                               const int3 *__restrict__ cell_index, const int3 resolution,
                               float *__restrict__ density, float *__restrict__ pressure,
                               const float smooth_len2, const float K_poly6, const float stiffness,
                               const unsigned int n, const int offset)
{
    PX_CUDA_LOOP(id, n)
    {
        auto rho = 0.f;
        auto i = id + offset;

        auto x_s = cell_index[i].x - 1;
        auto x_e = cell_index[i].x + 2;
        auto y_s = cell_index[i].y - 1;
        auto y_e = cell_index[i].y + 2;
        auto z_s = cell_index[i].z - 1;
        auto z_e = cell_index[i].z + 2;
        if (x_s < 0) x_s = 0;
        if (x_e > resolution.x) x_e = resolution.x;
        if (y_s < 0) y_s = 0;
        if (y_e > resolution.y) y_e = resolution.y;
        if (z_s < 0) z_s = 0;
        if (z_e > resolution.z) z_e = resolution.z;
        for (int ix = x_s; ix < x_e; ix++)
        {
            for (int iy = y_s; iy < y_e; iy++)
            {
                for (int iz = z_s; iz < z_e; iz++)
                {
                    auto index =
                            (ix * resolution.y + iy) * resolution.z + iz;


                    auto first = index > 0 ? bucket_start[index - 1] : 0;
                    auto last = bucket_start[index];
                    for (int neighbor = first; neighbor < last; neighbor++)
                    {
                        auto j = bucket_map[neighbor];
                        auto r = pos[i] - pos[j];
                        auto r2 = r.x*r.x+r.y*r.y+r.z*r.z;

                        if (r2 < smooth_len2)
                        {
                            auto rest_dist2 = smooth_len2 - r2;
                            rho = rho+mass[j]*rest_dist2*rest_dist2*rest_dist2;
                        }
                    }
                }
            }
        }
        rho *= K_poly6;
        auto pa =  stiffness * (rho - rest_density[i]);
        density[i] = rho;
        pressure[i] = pa > 0.f ? pa : 0.f;
    }
}
__global__
void cudaSPHAccleration(const float3 *__restrict__ pos, const float3 *__restrict__ vel,
                        const float *__restrict__ mass, const float *__restrict__ coef_viscosity,
                        const float *__restrict__ density, const float *__restrict__ pressure,
                        const int *__restrict__ bucket_map, const int *__restrict__ bucket_start, const int3 *__restrict__ cell_index, const int3 resolution,
                        float3 *__restrict__ acceleration, float *__restrict__ color,
                        const float smooth_len, const float smooth_len2,
                        const float3 gravity,
                        const float extra_coef_pressure,
                        const float extra_coef_viscosity,
                        const float neg_coef_surface_tension,
                        const float K_delta_spiky, const float K_delta_poly6, const float K_laplacian_poly6,
                        const unsigned int n, const int offset)
{
    PX_CUDA_LOOP(id, n)
    {
        auto i = id + offset;

        auto rho = density[i];
        auto p = pos[i];
        auto v = vel[i];
        float3 f_v, f_p, d_color;
        f_v.x = 0.f; f_p.x = 0.f; d_color.x = 0.f;
        f_v.y = 0.f; f_p.y = 0.f; d_color.y = 0.f;
        f_v.z = 0.f; f_p.z = 0.f; d_color.z = 0.f;
        auto d_color2 = 0.f;

        auto x_s = cell_index[i].x - 1;
        auto x_e = cell_index[i].x + 2;
        auto y_s = cell_index[i].y - 1;
        auto y_e = cell_index[i].y + 2;
        auto z_s = cell_index[i].z - 1;
        auto z_e = cell_index[i].z + 2;
        if (x_s < 0) x_s = 0;
        if (x_e > resolution.x) x_e = resolution.x;
        if (y_s < 0) y_s = 0;
        if (y_e > resolution.y) y_e = resolution.y;
        if (z_s < 0) z_s = 0;
        if (z_e > resolution.z) z_e = resolution.z;
        for (int ix = x_s; ix < x_e; ix++)
        {
            for (int iy = y_s; iy < y_e; iy++)
            {
                for (int iz = z_s; iz < z_e; iz++)
                {
                    auto index =
                            (ix * resolution.y + iy) * resolution.z + iz;
                    auto first = index > 0 ? bucket_start[index - 1] : 0;
                    auto last = bucket_start[index];
                    for (int neighbor = first; neighbor < last; neighbor++)
                    {
                        auto j = bucket_map[neighbor];
                        if (i == j) continue;

                        auto r = p - pos[j];
                        auto r2 = r.x*r.x+r.y*r.y+r.z*r.z;
                        if (r2 < smooth_len2)
                        {
                            auto dist = sqrtf(r2);
                            auto rest_len = smooth_len - dist;
                            auto rest_len2 = smooth_len2 - r2;
                            auto volume_rest_len = rest_len * mass[j]/density[j];

                            auto p_sum = pressure[i] + pressure[j];
//
                            if (p_sum != 0.f && dist != 0.f)
                                f_p = f_p + (p_sum*0.5f*volume_rest_len*rest_len/dist) * r;
                            f_v = f_v + (coef_viscosity[j]*volume_rest_len) * (vel[j] - v);
                            d_color = d_color + (volume_rest_len*rest_len2*rest_len2) * r;
                            d_color2 = d_color2 + volume_rest_len*rest_len2*(7.f*r2 - 3.f*smooth_len2);
                        }
                    }
                }
            }
        }

        d_color = K_delta_poly6  * d_color;
//        auto min = (abs(d_color.x) < abs(d_color.y) && abs(d_color.x) > 0.f) ? abs(d_color.x) : abs(d_color.y);
//        min = (min < abs(d_color.z) && min > 0.f) ? min : abs(d_color.z);
//        if (min > 0.f)
//        {
//            d_color.x = d_color.x/min;
//            d_color.y = d_color.y/min;
//            d_color.z = d_color.z/min;
            auto len_delta_color_sq = d_color.x*d_color.x + d_color.y*d_color.y + d_color.z*d_color.z;
            if (len_delta_color_sq > 0.f)
            {
                d_color = neg_coef_surface_tension*K_laplacian_poly6*d_color2/sqrtf(len_delta_color_sq) * d_color;
            }
//        }
//        color[i] = (d_color2*K_laplacian_poly6-1000)*1e-4f;
        // TODO surface tension visualization
        color[i] = 0.f;
        acceleration[i] = gravity + (d_color + K_delta_spiky*(extra_coef_pressure * f_p + extra_coef_viscosity * f_v)) / rho;
    }
}
__global__
void cudaSPHLeapfrog(float3 *__restrict__ pos, float3 *__restrict__ vel, float3 *__restrict__ vel_h, const float3 *__restrict__ acc,
                     const float3 lower_boundary, const float3 upper_boundary, const float particle_diameter,
                     const float dt,
                     const unsigned int n, const int offset)
{
    constexpr auto eps = 1e-6f;
    constexpr auto stiff = 5e4f;
    constexpr auto damp = 100.f;

#define BOUNDARY_CONSTRAINT(AXIS)                                               \
            {                                                                   \
                auto diff = particle_diameter - (p.AXIS - lower_boundary.AXIS); \
                if (diff > eps) a.AXIS = stiff * diff - damp * v.AXIS;          \
            }                                                                   \
            {                                                                   \
                auto diff = (upper_boundary.AXIS - p.AXIS) - particle_diameter; \
                if (diff < -eps) a.AXIS = stiff * diff - damp * v.AXIS;         \
            }

    PX_CUDA_LOOP(id, n)
    {
        auto i = id+offset;

        auto p = pos[i] + vel_h[i] * dt;
        auto v = vel[i];
        float3 a;
        a.x = 0.f;
        a.y = 0.f;
        a.z = 0.f;

        BOUNDARY_CONSTRAINT(x);
        BOUNDARY_CONSTRAINT(y);
        BOUNDARY_CONSTRAINT(z);

        auto dv = (acc[i]+ a)*dt;

        vel[i]   = vel_h[i] + dv*0.5f;
        vel_h[i] = vel_h[i] + dv;
        pos[i]   = pos[i] + vel_h[i]*dt;
    }
}


void scene::SPHScene::cudaInit(const float *mass, const float *rest_density, const float *coef_viscosity,
                               unsigned int capacity)
{
    if (cuda_buffer.streams == nullptr && cuda_buffer.n_streams > 0)
    {
        cuda_buffer.streams = new hipStream_t[cuda_buffer.n_streams];
        for (auto i = 0; i < cuda_buffer.n_streams; i++)
            PX_CUDA_CHECK(hipStreamCreateWithFlags(&cuda_buffer.streams[i], hipStreamNonBlocking));
    }
    if (capacity > cuda_buffer.capacity)
    {
        PX_CUDA_CHECK(hipMalloc(&cuda_buffer.acceleration, sizeof(float3)*capacity));
        PX_CUDA_CHECK(hipMalloc(&cuda_buffer.density, sizeof(float)*capacity));
        PX_CUDA_CHECK(hipMalloc(&cuda_buffer.pressure, sizeof(float)*capacity));
        PX_CUDA_CHECK(hipMalloc(&cuda_buffer.grid_index, sizeof(int)*capacity));
        PX_CUDA_CHECK(hipMalloc(&cuda_buffer.cell_index, sizeof(int3)*capacity));
        PX_CUDA_CHECK(hipMalloc(&cuda_buffer.bucket, sizeof(int)*capacity));
        PX_CUDA_CHECK(hipMalloc(&cuda_buffer.bucket_map, sizeof(int)*capacity));

        PX_CUDA_CHECK(hipMalloc(&cuda_buffer.grid_counter, sizeof(int)*n_grids));
        PX_CUDA_CHECK(hipMalloc(&cuda_buffer.bucket_start, sizeof(int)*n_grids));

        if (cuda_buffer.capacity > 0)
        {
            void *mass = nullptr;
            void *rest_density = nullptr;
            void *coef_viscosity = nullptr;
            void *velocity = nullptr;
            void *velocity_h = nullptr;
            PX_CUDA_CHECK(hipMalloc(&mass, sizeof(float)*capacity));
            PX_CUDA_CHECK(hipMalloc(&rest_density, sizeof(float)*capacity));
            PX_CUDA_CHECK(hipMalloc(&coef_viscosity, sizeof(float)*capacity));
            PX_CUDA_CHECK(hipMalloc(&velocity, sizeof(float3)*capacity));
            PX_CUDA_CHECK(hipMalloc(&velocity_h, sizeof(float3)*capacity));

            PX_CUDA_CHECK(hipMemcpy(velocity, cuda_buffer.mass, sizeof(float3)*cuda_buffer.capacity, hipMemcpyDeviceToDevice));
            PX_CUDA_CHECK(hipMemcpy(velocity, cuda_buffer.velocity, sizeof(float3)*cuda_buffer.capacity, hipMemcpyDeviceToDevice));
            PX_CUDA_CHECK(hipMemcpy(velocity_h, cuda_buffer.velocity_h, sizeof(float3)*cuda_buffer.capacity, hipMemcpyDeviceToDevice));

            PX_CUDA_CHECK(hipFree(cuda_buffer.mass));
            PX_CUDA_CHECK(hipFree(cuda_buffer.rest_density));
            PX_CUDA_CHECK(hipFree(cuda_buffer.coef_viscosity));
            PX_CUDA_CHECK(hipFree(cuda_buffer.velocity));
            PX_CUDA_CHECK(hipFree(cuda_buffer.velocity_h));

            cuda_buffer.mass = reinterpret_cast<float *>(mass);
            cuda_buffer.rest_density = reinterpret_cast<float *>(rest_density);
            cuda_buffer.coef_viscosity = reinterpret_cast<float *>(coef_viscosity);
            cuda_buffer.velocity = reinterpret_cast<float3 *>(velocity);
            cuda_buffer.velocity_h = reinterpret_cast<float3 *>(velocity_h);
        }
        else
        {
            PX_CUDA_CHECK(hipFree(cuda_buffer.mass));
            PX_CUDA_CHECK(hipFree(cuda_buffer.rest_density));
            PX_CUDA_CHECK(hipFree(cuda_buffer.coef_viscosity));
            PX_CUDA_CHECK(hipFree(cuda_buffer.velocity));
            PX_CUDA_CHECK(hipFree(cuda_buffer.velocity_h));

            PX_CUDA_CHECK(hipMalloc(&cuda_buffer.mass, sizeof(float)*capacity));
            PX_CUDA_CHECK(hipMalloc(&cuda_buffer.rest_density, sizeof(float)*capacity));
            PX_CUDA_CHECK(hipMalloc(&cuda_buffer.coef_viscosity, sizeof(float)*capacity));
            PX_CUDA_CHECK(hipMalloc(&cuda_buffer.velocity, sizeof(float3)*capacity));
            PX_CUDA_CHECK(hipMalloc(&cuda_buffer.velocity_h, sizeof(float3)*capacity));

            PX_CUDA_CHECK(hipMemset(cuda_buffer.velocity, 0, sizeof(float3)*capacity));
            PX_CUDA_CHECK(hipMemset(cuda_buffer.velocity_h, 0, sizeof(float3)*capacity));
        }
        cuda_buffer.capacity = capacity;
    }
    PX_CUDA_CHECK(hipMemcpy(cuda_buffer.mass, mass, sizeof(float)*capacity, hipMemcpyHostToDevice));
    PX_CUDA_CHECK(hipMemcpy(cuda_buffer.rest_density, rest_density, sizeof(float)*capacity, hipMemcpyHostToDevice));
    PX_CUDA_CHECK(hipMemcpy(cuda_buffer.coef_viscosity, coef_viscosity, sizeof(float)*capacity, hipMemcpyHostToDevice));
}

void scene::SPHScene::cudaAppend(float3* pos_buffer, unsigned int offset,
                                 const float3* pos, const float3* vel, const float* mass,
                                 const float* rest_density, const float *coef_viscosity,
                                 int n)
{
    PX_CUDA_CHECK(hipMemcpy(pos_buffer + offset, pos, sizeof(float3)*n, hipMemcpyHostToDevice));
    PX_CUDA_CHECK(hipMemcpy(cuda_buffer.velocity + offset, vel, sizeof(float3)*n, hipMemcpyHostToDevice));
    PX_CUDA_CHECK(hipMemcpy(cuda_buffer.velocity_h + offset, vel, sizeof(float3)*n, hipMemcpyHostToDevice));
    PX_CUDA_CHECK(hipMemcpy(cuda_buffer.mass + offset, mass, sizeof(float)*n, hipMemcpyHostToDevice));
    PX_CUDA_CHECK(hipMemcpy(cuda_buffer.rest_density + offset, rest_density, sizeof(float)*n, hipMemcpyHostToDevice));
    PX_CUDA_CHECK(hipMemcpy(cuda_buffer.coef_viscosity + offset, coef_viscosity, sizeof(float)*n, hipMemcpyHostToDevice));
}

void scene::SPHScene::cudaDrop()
{
    PX_CUDA_CHECK(hipFree(cuda_buffer.mass));
    PX_CUDA_CHECK(hipFree(cuda_buffer.rest_density));
    PX_CUDA_CHECK(hipFree(cuda_buffer.coef_viscosity));
    PX_CUDA_CHECK(hipFree(cuda_buffer.velocity));
    PX_CUDA_CHECK(hipFree(cuda_buffer.velocity_h));
    PX_CUDA_CHECK(hipFree(cuda_buffer.acceleration));
    PX_CUDA_CHECK(hipFree(cuda_buffer.density));
    PX_CUDA_CHECK(hipFree(cuda_buffer.pressure));
    PX_CUDA_CHECK(hipFree(cuda_buffer.grid_index));
    PX_CUDA_CHECK(hipFree(cuda_buffer.cell_index));
    PX_CUDA_CHECK(hipFree(cuda_buffer.bucket));
    PX_CUDA_CHECK(hipFree(cuda_buffer.bucket_map));

    cuda_buffer.mass = nullptr;
    cuda_buffer.rest_density = nullptr;
    cuda_buffer.coef_viscosity = nullptr;
    cuda_buffer.velocity = nullptr;
    cuda_buffer.velocity_h = nullptr;
    cuda_buffer.acceleration = nullptr;
    cuda_buffer.density = nullptr;
    cuda_buffer.pressure = nullptr;
    cuda_buffer.grid_index = nullptr;
    cuda_buffer.cell_index = nullptr;
    cuda_buffer.bucket = nullptr;
    cuda_buffer.bucket_map = nullptr;

    PX_CUDA_CHECK(hipFree(cuda_buffer.grid_counter));
    PX_CUDA_CHECK(hipFree(cuda_buffer.bucket_start));
    cuda_buffer.bucket_start = nullptr;
    cuda_buffer.grid_counter = nullptr;

    if (cuda_buffer.streams != nullptr)
    {
        for (int i = 0; i < cuda_buffer.n_streams; i++)
            PX_CUDA_CHECK(hipStreamDestroy(cuda_buffer.streams[i]));
        delete cuda_buffer.streams;
        cuda_buffer.streams = nullptr;
    }

    cuda_buffer.capacity = 0;
}

void scene::SPHScene::cudaUpdate(float3 *pos, float *color, float dt)
{
    const static float3 zero3 = make_float3(0.f, 0.f, 0.f);

    // see http://on-demand.gputechconf.com/gtc/2014/presentations/S4117-fast-fixed-radius-nearest-neighbor-gpu.pdf
    // this implementation is based on the above doc, but not exactly the same

    auto n_particles = particles();
    if (n_particles < 1) return;

    auto n_particles_per_stream = cuda_buffer.n_streams == 0 ? 0 : n_particles / cuda_buffer.n_streams;
    auto remaining = n_particles - n_particles_per_stream*cuda_buffer.n_streams;
    auto blocks = cuda::blocks(n_particles);
    auto blocks_per_stream = cuda::blocks(n_particles_per_stream);
    auto blocks_remaining = cuda::blocks(remaining);


    int offset;
#define STREAM_CALL(FUNC, args...)                                                              \
    {                                                                                           \
        offset = 0;                                                                             \
        for (auto i = 0; i < cuda_buffer.n_streams; i++)                                        \
        {                                                                                       \
            FUNC<<<blocks_per_stream, PX_CUDA_THREADS_PER_BLOCK, 0, cuda_buffer.streams[i]>>>(  \
                    args,                                                                       \
                    n_particles_per_stream, offset                                              \
            );                                                                                  \
            offset += n_particles_per_stream;                                                   \
        }                                                                                       \
        if (remaining > 0)                                                                      \
            FUNC<<<blocks_remaining, PX_CUDA_THREADS_PER_BLOCK, 0>>>(                           \
                    args,                                                                       \
                    remaining, offset                                                           \
            );                                                                                  \
        PX_CUDA_CHECK(hipDeviceSynchronize());                                                 \
    };

//    TIC(0);
    cudaSPHClearCounter<<<cuda::blocks(n_grids), PX_CUDA_THREADS_PER_BLOCK>>>(
            cuda_buffer.grid_counter, n_grids
    );
//    hipStream_t s;
//    hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
//    hipMemsetAsync(cuda_buffer.grid_counter, 0, sizeof(int)*n_grids, s);
//    PX_CUDA_CHECK(hipDeviceSynchronize());
//    TOC(0)


//    TIC(1);
    STREAM_CALL(cudaSPHPosition2Grid,
                pos, cuda_buffer.grid_index, cuda_buffer.cell_index,
                lower_boundary, smooth_len, resolution);
//    TOC(1);

//    TIC(2);
    cudaSPHGridCount<<<blocks, PX_CUDA_THREADS_PER_BLOCK>>>(
            cuda_buffer.grid_index, cuda_buffer.grid_counter, n_particles
    );
//    PX_CUDA_CHECK(hipDeviceSynchronize());
//    TOC(2)
//    TIC(3);
    thrust::device_ptr<int> grid_ptr(cuda_buffer.grid_counter);
    thrust::device_vector<int> counter(n_grids);
    thrust::exclusive_scan(grid_ptr, grid_ptr+n_grids, counter.begin());
    auto pref_sum = thrust::raw_pointer_cast(&counter[0]);
//    TOC(3)
//    TIC(4)
    cudaSPHSort<<<blocks, PX_CUDA_THREADS_PER_BLOCK>>>(
            pref_sum, cuda_buffer.grid_index, cuda_buffer.bucket,
            cuda_buffer.bucket_map, n_particles
    );
//    PX_CUDA_CHECK(hipDeviceSynchronize());
//    TOC(4)
//    TIC(5)
    cudaSPHMakeBins<<<blocks, PX_CUDA_THREADS_PER_BLOCK>>>(
           cuda_buffer.bucket, cuda_buffer.bucket_start,
           n_grids, n_particles
    );
    PX_CUDA_CHECK(hipDeviceSynchronize());
//    TOC(5)
//
//
//    TIC(6)
    STREAM_CALL(cudaSPHDensityAndPressure,
                pos, cuda_buffer.mass, cuda_buffer.rest_density,
                cuda_buffer.bucket_map, cuda_buffer.bucket_start, cuda_buffer.cell_index, resolution,
                cuda_buffer.density, cuda_buffer.pressure,
                smooth_len2, K_poly6, stiffness);
//    TOC(6)
//    TIC(7)
    STREAM_CALL(cudaSPHAccleration,
                pos, cuda_buffer.velocity, cuda_buffer.mass, cuda_buffer.coef_viscosity,
                cuda_buffer.density, cuda_buffer.pressure,
                cuda_buffer.bucket_map, cuda_buffer.bucket_start, cuda_buffer.cell_index, resolution,
                cuda_buffer.acceleration, color,
                smooth_len, smooth_len2,
                (scene_param.gravity_force ? gravity : zero3),
                (scene_param.pressure_force ? 1.f : 0.f),
                (scene_param.viscosity_force ? 1.f : 0.f),
                (scene_param.surface_tension ? neg_coef_surface_tension : 0.f),
                K_delta_spiky, K_delta_poly6, K_laplacian_poly6);
//    TOC(7)
//    TIC(8)
    STREAM_CALL(cudaSPHLeapfrog,
                pos, cuda_buffer.velocity, cuda_buffer.velocity_h, cuda_buffer.acceleration,
                lower_boundary, upper_boundary, particle_diameter,
                dt);
//    TOC(8)
}

